#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
using namespace std;
#include <stdio.h>
#include <algorithm>
#include <ctime>
#include <cstdlib> 


const int sizePoint = 700; 		//Количество точек
const int sizeIndividum = 1000;		//Количество индивидумов
const int MutationProbability = 10;	//Вероятность мутации в процентах
const float MutationDispersion = 5.0f;	//Дисперсия мутации
const int Polynom = 3;			//Полином
const float randMaxCount = 20.0f;	//Верхняя граница заполнения точек
const int maxGeneration = 30;		//Число поколений

__global__ void errorsKernel(float* points, 
			     float* individs, 
			     float* errors, 
			     int Polynom, 
			     int sizePoint)					//  Вычисление ошибки GPU
{

	int id = threadIdx.x;							//Поток
	float ans = 0;
	int x = 1;
	for (int i = 0; i < sizePoint; i++)
	{
		for (int j = 0; j < Polynom; j++)
		{
			for (int k = 0; k < j; k++)
			{
				x *= i;
			}
			x *= individs[id * Polynom + j];
			ans += x;
			x = 1;
		}

		ans = points[i] - ans;
		errors[id] += sqrt(ans * ans);
		ans = 0;
	}
}


void testErrorsKernel(float* points, 
		      float* individs, 
		      float* errors, 
		      int Polynom, 
		      int sizePoint, 
		      int sizeIndividum)		// вычисление ошибки на CPU
{
	for (int id = 0; id < sizeIndividum; id++)
	{
		float ans = 0.0f;
		errors[id] = 0.0f;
		int x = 0;
		for (int i = 0; i < sizePoint; i++)
		{
			for (int j = 0; j < Polynom; j++)
			{
				x = pow(i, j);
				x *= individs[id * Polynom + j];
				ans += x;
				x = 0;
			}

			ans = points[i] - ans;
			errors[id] += sqrt(ans * ans);
			ans = 0;
		}
	}
}

float RandomFloat(float a, float b) {
	float random = ((float)rand()) / (float)RAND_MAX;
	float diff = b - a;
	float r = random * diff;
	return a + r;
}

void cpu() {
	float* pointsH = new float[sizePoint]; 
	for (int i = 0; i < sizePoint; i++)							// создание случайного набора точек
	{
		pointsH[i] = RandomFloat(0, randMaxCount);
	}

	float* individumsH = new float[sizeIndividum * Polynom];
	for (int i = 0; i < sizeIndividum * Polynom; i++)			// создание первого поколения
	{
		individumsH[i] = RandomFloat(0, randMaxCount);			
	}

	float* errorsH = new float[sizeIndividum];
	for (int i = 0; i < sizeIndividum; i++)
	{
		errorsH[i] = 1000;
	}

	unsigned int start_time_cpu = clock(); 

	for (int generation = 0; generation < maxGeneration; generation++)
	{
		testErrorsKernel(pointsH, 
				 individumsH, 
				 errorsH, 
				 Polynom, 
				 sizePoint, 
				 sizeIndividum);

		float* errorsCrossOver = new float[sizeIndividum];		//скрещивание

		for (size_t i = 0; i != sizeIndividum; ++i)
			errorsCrossOver[i] = errorsH[i];
		sort(errorsCrossOver, errorsCrossOver + sizeIndividum);

		int merodianCrossOvering = sizeIndividum / 2;
		float merodianErrorCrossOvering = errorsCrossOver[merodianCrossOvering];
		float* theBestInd = new float[Polynom];

		for (size_t i = 0; i < sizeIndividum; i++)			// смена поколения (30 раз)
		{								// Особи с худшими значениями зануляются
			if (merodianErrorCrossOvering < errorsH[i]) {		// Лучшие заполняют их своими значениями	
				for (size_t j = 0; j < Polynom; j++)		// Лучшими считаются те, у которых значение ошибки меньше 
				{
					individumsH[i * Polynom + j] = 0;
				}
			}
			if (errorsH[i] == errorsCrossOver[0]) {
				for (int j = 0; j < Polynom; j++)
				{
					theBestInd[j] = individumsH[i * Polynom + j];
				}
			}
		}

		printf("Error = %f\n", errorsCrossOver[0]);

		for (int i = 0; i < sizeIndividum * Polynom; i++)
		{
			if (individumsH[i] == 0) {
				individumsH[i] = theBestInd[rand() % Polynom];
			}

			if (MutationProbability > (rand() % 100 + 1)) {
				individumsH[i] += RandomFloat(-MutationDispersion, MutationDispersion);
			}
		}
	}
	unsigned int end_time_cpu = clock(); 
	unsigned int search_time_cpu = end_time_cpu - start_time_cpu;
	printf("Time CPU = %i\n", search_time_cpu);
}

void gpu() {
	float* pointsH = new float[sizePoint];
	for (int i = 0; i < sizePoint; i++)				// Создаем случайный набор точек
	{  
		pointsH[i] = RandomFloat(0, randMaxCount);
	}

	float* individumsH = new float[sizeIndividum * Polynom];
	for (int i = 0; i < sizeIndividum * Polynom; i++)	// Создаем первое поколение
	{
		individumsH[i] = RandomFloat(0, randMaxCount);
	}

	float* errorsH = new float[sizeIndividum];
	for (int i = 0; i < sizeIndividum; i++)
	{
		errorsH[i] = 1000;
	}

	unsigned int start_time_gpu = clock();
	float* pointsD = NULL;
	float* individumsD = NULL;
	float* errorsD = NULL;

	for (int generation = 0; generation < maxGeneration; generation++)
	{

		int sizeIndividumBytes = sizeIndividum * Polynom * sizeof(float);
		int sizePointBytes = sizePoint * sizeof(float);
		//Выделяем память
		hipMalloc((void**)&pointsD, sizePointBytes);
		hipMalloc((void**)&individumsD, sizeIndividumBytes * Polynom);
		hipMalloc((void**)&errorsD, sizeIndividum * sizeof(float));
		//Копируем ввод
		hipMemcpy(pointsD, pointsH, sizePointBytes, hipMemcpyHostToDevice);
		hipMemcpy(individumsD, individumsH, sizeIndividumBytes, hipMemcpyHostToDevice);
		hipMemcpy(errorsD, errorsH, sizeIndividumBytes, hipMemcpyHostToDevice);

		errorsKernel << <1, sizeIndividum >> > (pointsD, individumsD, errorsD, Polynom, sizePoint);

		hipMemcpy(errorsH, errorsD, sizeIndividum * sizeof(float), hipMemcpyDeviceToHost);

		//----------------------
		float* errorsCrossOver = new float[sizeIndividum];

		for (size_t i = 0; i != sizeIndividum; ++i)
			errorsCrossOver[i] = errorsH[i];
		sort(errorsCrossOver, errorsCrossOver + sizeIndividum);
		printf("Error = %f\n", errorsCrossOver[0]);
		int merodianCrossOvering = sizeIndividum / 2;
		float merodianErrorCrossOvering = errorsCrossOver[merodianCrossOvering];
		float* theBestInd = new float[Polynom];

		for (size_t i = 0; i < sizeIndividum; i++)				// Смена поколения
		{									// Особи с худшими значениями зануляются
			if (merodianErrorCrossOvering < errorsH[i]) {			// Лучшие заполняют их своими значениями
				for (size_t j = 0; j < Polynom; j++)			// Лучшими считаются те, у которых значение ошибки меньше			
				{
					individumsH[i * Polynom + j] = 0;
				}
			}
			if (errorsH[i] == errorsCrossOver[0]) {
				for (int j = 0; j < Polynom; j++)
				{
					theBestInd[j] = individumsH[i * Polynom + j];
				}
			}
		}

		for (int i = 0; i < sizeIndividum * Polynom; i++)
		{
			if (individumsH[i] == 0) {
				individumsH[i] = theBestInd[rand() % Polynom];
			}

			if (MutationProbability > (rand() % 100 + 1)) {
				individumsH[i] += RandomFloat(-MutationDispersion, MutationDispersion);
			}
		}
	}
	unsigned int end_time_gpu = clock();
	unsigned int search_time_gpu = end_time_gpu - start_time_gpu;

	printf("Time GPU = %i\n", search_time_gpu);
	
	hipFree(pointsD);
	hipFree(individumsD);
	hipFree(errorsD);

	delete pointsH;
	delete individumsH;
	delete errorsH;
}

int main()
{
	cpu();
	gpu();
	system("pause");
	return 0;
}
